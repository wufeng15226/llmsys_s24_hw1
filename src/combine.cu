#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define BLOCK_DIM 1024
#define MAX_DIMS 10
#define TILE 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

#define NO_STRIDES
#define ZTY_DEBUG

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        // BEGIN ASSIGN1_0
        return x;
        // END ASSIGN1_0
      }
      case TANH: {
        // BEGIN ASSIGN1_0
        return x;
        // END ASSIGN1_0
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
  /**
   * Converts a multidimensional tensor index into a single-dimensional position in storage
   * based on strides.
   * Args:
   *    index: index tuple of ints
   *    strides: tensor strides
   *    num_dims: number of dimensions in the tensor, e.g. shape/strides of [2, 3, 4] has 3 dimensions
   * 
   * Returns:
   *    int - position in storage
  */
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
  /**
   * Convert an ordinal to an index in the shape. Should ensure that enumerating position 0 ... size of 
   * a tensor produces every index exactly once. It may not be the inverse of index_to_position.
   * Args:
   *    ordinal: ordinal position to convert
   *    shape: tensor shape
   *    out_index: return index corresponding to position
   *    num_dims: number of dimensions in the tensor
   * 
   * Returns:
   *    None (Fills in out_index) 
  */
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
  /**
   * Convert a big_index into big_shape to a smaller out_index into shape following broadcasting rules. 
   * In this case it may be larger or with more dimensions than the shape given. 
   * Additional dimensions may need to be mapped to 0 or removed.
   * 
   * Args:
   *    big_index: multidimensional index of bigger tensor
   *    big_shape: tensor shape of bigger tensor
   *    nums_big_dims: number of dimensions in bigger tensor
   *    out_index: multidimensional index of smaller tensor
   *    shape: tensor shape of smaller tensor  
   *    num_dims: number of dimensions in smaller tensor
   * 
   * Returns:
   *    None (Fills in out_index) 
  */
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}

__device__ void MatrixMultiplyNoStridesKernel(
  int m,
  int n,
  int k,
  float* a,
  float* b,
  float* out
){
  // (m,n) x (n,k)
  // careful about gridDims, i/x for row, j/y for col
  
  // no shared memory
  // int j = blockIdx.y * blockDim.y + threadIdx.y;
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // if(i<m&&j<k){
  //   float sum = 0;
  //   for(int ii=0;ii<n;++ii){
  //     sum += a[i*n+ii]*b[ii*k+j];
  //   }
  //   out[i*k+j] = sum;
  // }

  // #ifdef ZTY_DEBUG
  //   printf("%d %d %d %d %f\n", blockIdx.y, threadIdx.y, blockIdx.x, threadIdx.x, out[i*k+j]);
  // #endif

  __shared__ float a_shared[TILE][TILE];
  __shared__ float b_shared[TILE][TILE];

  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int tailj = threadIdx.y;
  int taili = threadIdx.x;
  float sum = 0;
  for(int stride=0;stride<n;stride+=TILE){
    if(i<m&&(stride+tailj)<n){
      a_shared[taili][tailj] = a[i*n+stride+tailj];
    }else{
      a_shared[taili][tailj] = 0;
    }
    if((stride+taili)<n&&j<k){
      b_shared[taili][tailj] = b[(stride+taili)*k+j];
    }else{
      b_shared[taili][tailj] = 0;
    }
    __syncthreads();

    for(int ii=0;ii<TILE;++ii){
      sum += a_shared[taili][ii]*b_shared[ii][tailj];
    }
    __syncthreads();
  }
  if(i<m&&j<k) out[i*k+j] = sum;
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix. Matrix a and b are both in a batch 
   * format, with shape [batch_size, m, n], [batch_size, n, p].
   * Requirements:
   * - All data must be first moved to shared memory.
   * - Only read each cell in a and b once.
   * - Only write to global memory once per kernel.
   * There is guarantee that a_shape[0] == b_shape[0], a_shape[2] == b_shape[1], 
   * and out_shape[0] == a_shape[0], out_shape[1] == b_shape[1]
   * 
   * Args:
   *   out: compact 1D array of size batch_size x m x p to write the output to
   *   out_shape: shape of the output array
   *   out_strides: strides of the output array
   *   a_storage: compact 1D array of size batch_size x m x n
   *   a_shape: shape of the a array
   *   a_strides: strides of the a array
   *   b_storage: comapct 2D array of size batch_size x n x p
   *   b_shape: shape of the b array
   *   b_strides: strides of the b array
   * 
   * Returns:
   *   None (Fills in out array)
   */
#ifdef NO_STRIDES
  int batch = blockIdx.z;
  int m = a_shape[1], n = a_shape[2], p = b_shape[2];
  MatrixMultiplyNoStridesKernel(m, n, p, a_storage+a_strides[0]*batch, b_storage+b_strides[0]*batch, out+out_strides[0]*batch);
#else
    __shared__ float a_shared[TILE][TILE];
    __shared__ float b_shared[TILE][TILE];

    // In each block, we will compute a batch of the output matrix
    // All the threads in the block will work together to compute this batch
    int batch = blockIdx.z;

    /// BEGIN ASSIGN1_2
    // Hints:
    // 1. Compute the row and column of the output matrix this block will compute
    // 2. Compute the position in the output array that this thread will write to
    // 3. Iterate over tiles of the two input matrices, read the data into shared memory
    // 4. Synchronize to make sure the data is available to all threads
    // 5. Compute the output tile for this thread block
    // 6. Synchronize to make sure all threads are done computing the output tile for (row, col)
    // 7. Write the output to global memory

    int ix = threadIdx.x;
    int iy = threadIdx.y;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // (m,n) x (n,p)
    int m = a_shape[1], n = a_shape[2], p = b_shape[2];

    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

    out_index[0] = batch;
    out_index[1] = x;
    out_index[2] = y;

    int out_pos = index_to_position(out_index, out_strides, 3); 
    float sum = 0;
    for (int i = 0; i < (n+TILE-1)/TILE; i++) {
      a_index[0] = batch;
      a_index[1] = x;
      a_index[2] = i*TILE + iy;
      b_index[0] = batch;
      b_index[1] = i*TILE + ix;
      b_index[2] = y;

      if (a_index[1] < m && a_index[2] < n) {
        int a_pos = index_to_position(a_index, a_strides, 3);
        a_shared[ix][iy] = a_storage[a_pos];
      }
      else {
        a_shared[ix][iy] = 0;
      }
      if (b_index[1] < n && b_index[2] < p) {
        int b_pos = index_to_position(b_index, b_strides, 3);
        b_shared[ix][iy] = b_storage[b_pos];
      }
      else {
        b_shared[ix][iy] = 0;
      }
      __syncthreads();

      for (int j = 0; j < TILE; j++) {
        if (i + j < n) {
          sum += a_shared[ix][j] * b_shared[j][iy];
        }
      }
      __syncthreads();
    }

    if (x < out_shape[1] && y < out_shape[2]) {
      out[out_pos] = sum;
    }
    /// END ASSIGN1_2
#endif
}


__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
  /**
   * Map function. Apply a unary function to each element of the input array and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   * 
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   * 
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  in_storage: compact 1D array of size in_size
   *  in_shape: shape of the input array
   *  in_strides: strides of the input array
   *  shape_size: number of dimensions in the input and output arrays, assume dimensions are the same
   *  fn_id: id of the function to apply to each element of the input array
   * 
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];
    
    /// BEGIN ASSIGN1_2
    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    // 2. Convert the position to the out_index according to out_shape
    to_index(ix, out_shape, out_index, shape_size);
    // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
    broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
    // 4. Calculate the position of element in in_array according to in_index and in_strides
    int in_pos = index_to_position(in_index, in_strides, shape_size);
    // 5. Calculate the position of element in out_array according to out_index and out_strides
    int out_pos = index_to_position(out_index, out_strides, shape_size);
    // 6. Apply the unary function to the input element and write the output to the out memory
    out[out_pos] = fn(fn_id, in_storage[in_pos]);
    /// END ASSIGN1_2
}


__global__ void reduceKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
  /**
   * Reduce function. Apply a reduce function to elements of the input array a and store the result in the output array.
   * Optimization: 
   * Parallelize over the reduction operation. Each kernel performs one reduction.
   * e.g. a = [[1, 2, 3], [4, 5, 6]], kernel0 computes reduce([1, 2, 3]), kernel1 computes reduce([4, 5, 6]).
   * 
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * 
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  reduce_dim: dimension to reduce on
   *  reduce_value: initial value for the reduction
   *  shape_size: number of dimensions in the input & output array, assert dimensions are the same
   *  fn_id: id of the reduce function, currently only support add, multiply, and max
   *  
   * 
   * Returns:
   *  None (Fills in out array)
   */

    // __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
    int out_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    // 1. Define the position of the output element that this thread or this block will write to
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    // 2. Convert the out_pos to the out_index according to out_shape
    to_index(ix, out_shape, out_index, shape_size);
    // 3. Initialize the reduce_value to the output element
    float rst = reduce_value;
    // // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
    for(int i=0;i<a_shape[reduce_dim];++i){
      out_index[reduce_dim] = i;
      int a_pos = index_to_position(out_index, a_strides, shape_size);
      rst = fn(fn_id, rst, a_storage[a_pos]);
    }
    // 5. Write the reduced value to out memory
    out[ix] = rst;

    /// END ASSIGN1_2
}

__global__ void zipKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
  /**
   * Zip function. Apply a binary function to elements of the input array a & b and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   * 
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   * 
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  out_shape_size: number of dimensions in the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  a_shape_size: number of dimensions in the input array
   *  b_storage: compact 1D array of size in_size
   *  b_shape: shape of the input array
   *  b_strides: strides of the input array
   *  b_shape_size: number of dimensions in the input array
   *  fn_id: id of the function to apply to each element of the a & b array
   *  
   * 
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    // 2. Convert the position to the out_index according to out_shape
    // 3. Calculate the position of element in out_array according to out_index and out_strides
    // 4. Broadcast the out_index to the a_index according to a_shape
    // 5. Calculate the position of element in a_array according to a_index and a_strides
    // 6. Broadcast the out_index to the b_index according to b_shape
    // 7.Calculate the position of element in b_array according to b_index and b_strides
    // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
    
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    to_index(ix, out_shape, out_index, out_shape_size);
    int out_pos = index_to_position(out_index, out_strides, out_shape_size);
    broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
    int a_pos = index_to_position(a_index, a_strides, a_shape_size);
    broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
    int b_pos = index_to_position(b_index, b_strides, b_shape_size);
    out[out_pos] = fn(fn_id, a_storage[a_pos], b_storage[b_pos]);

    /// END ASSIGN1_2
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {
    float *d_out, *d_in;
    // Allocate device memory
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    // Copy data from CPU(host) to GPU(device)
    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {
    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    // Allocate device memory
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}
